#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <chrono>
#include <vector>
#include "kernels.cuh"

void runBenchmark(int N, int blockSize) {
    size_t bytes = N * N * sizeof(float);

    // Allocate host memory
    float* h_A = (float*)malloc(bytes);
    float* h_B = (float*)malloc(bytes);
    float* h_C = (float*)malloc(bytes);

    // Initialize matrices
    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1.0f + (float)(rand() % 100) / 100.0f;
        h_B[i] = 2.0f + (float)(rand() % 100) / 100.0f;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    // Conditional compilation for different versions
    #ifdef KERNEL_V1
        // Warmup
        runV1Baseline(d_A, d_B, d_C, N, blockSize);
        
        auto start = std::chrono::high_resolution_clock::now();
        runV1Baseline(d_A, d_B, d_C, N, blockSize);
        auto end = std::chrono::high_resolution_clock::now();
        
        double time_ms = std::chrono::duration<double, std::milli>(end - start).count();
        double gflops = 2.0 * N * N * N / (time_ms / 1000.0) / 1e9;
        printPerformanceRow(N, blockSize, time_ms, gflops);
        
    #elif defined(KERNEL_V2)
        // Warmup
        runV2LoopUnroll(d_A, d_B, d_C, N, blockSize);
        
        auto start = std::chrono::high_resolution_clock::now();
        runV2LoopUnroll(d_A, d_B, d_C, N, blockSize);
        auto end = std::chrono::high_resolution_clock::now();
        
        double time_ms = std::chrono::duration<double, std::milli>(end - start).count();
        double gflops = 2.0 * N * N * N / (time_ms / 1000.0) / 1e9;
        printPerformanceRow(N, blockSize, time_ms, gflops);
        
    #elif defined(KERNEL_V3)
        // Warmup
        runV3SharedMemory(d_A, d_B, d_C, N, blockSize);
        
        auto start = std::chrono::high_resolution_clock::now();
        runV3SharedMemory(d_A, d_B, d_C, N, blockSize);
        auto end = std::chrono::high_resolution_clock::now();
        
        double time_ms = std::chrono::duration<double, std::milli>(end - start).count();
        double gflops = 2.0 * N * N * N / (time_ms / 1000.0) / 1e9;
        printPerformanceRow(N, blockSize, time_ms, gflops);
        
    #elif defined(KERNEL_V4)
        // Warmup
        runV4ThreadCoarsening(d_A, d_B, d_C, N, blockSize);
        
        auto start = std::chrono::high_resolution_clock::now();
        runV4ThreadCoarsening(d_A, d_B, d_C, N, blockSize);
        auto end = std::chrono::high_resolution_clock::now();
        
        double time_ms = std::chrono::duration<double, std::milli>(end - start).count();
        double gflops = 2.0 * N * N * N / (time_ms / 1000.0) / 1e9;
        printPerformanceRow(N, blockSize, time_ms, gflops);
        
    #elif defined(KERNEL_V5)
        // Warmup
        runV5Privatization(d_A, d_B, d_C, N, blockSize);
        
        auto start = std::chrono::high_resolution_clock::now();
        runV5Privatization(d_A, d_B, d_C, N, blockSize);
        auto end = std::chrono::high_resolution_clock::now();
        
        double time_ms = std::chrono::duration<double, std::milli>(end - start).count();
        double gflops = 2.0 * N * N * N / (time_ms / 1000.0) / 1e9;
        printPerformanceRow(N, blockSize, time_ms, gflops);
        
    #elif defined(KERNEL_V6)
        double time_ms, gflops;
        runV6CuBLAS(d_A, d_B, d_C, N, time_ms, gflops);
        printPerformanceRow(N, blockSize, time_ms, gflops);
        
    #else
        printf("No kernel version specified! Use -DKERNEL_V1, -DKERNEL_V2, etc.\n");
        return;
    #endif

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
}

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        printf("No CUDA devices found!\n");
        return -1;
    }

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("CUDA Device: %s\n", prop.name);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);

    // Print version title once at the beginning
    #ifdef KERNEL_V1
        printVersionTitle("V1 Baseline Kernel");
    #elif defined(KERNEL_V2)
        printVersionTitle("V2 Loop Unroll Kernel");
    #elif defined(KERNEL_V3)
        printVersionTitle("V3 Shared Memory Kernel");
    #elif defined(KERNEL_V4)
        printVersionTitle("V4 Thread Coarsening Kernel");
    #elif defined(KERNEL_V5)
        printVersionTitle("V5 Privatization Kernel");
    #elif defined(KERNEL_V6)
        printVersionTitle("V6 cuBLAS Kernel");
    #endif

    printPerformanceHeader();

    std::vector<int> matrixSizes = {512, 1024, 2048};
    std::vector<int> blockSizes = {16, 32};

    for (int N : matrixSizes) {
        for (int blockSize : blockSizes) {
            if (blockSize * blockSize <= prop.maxThreadsPerBlock) {
                runBenchmark(N, blockSize);
            }
        }
    }

    printTableFooter();

    return 0;
}