#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "kernels.cuh"

#define TILE_SIZE 16
#define REG_TILE_SIZE 4

// V5: Privatization kernel - register tiling with thread coarsening
__global__ void V5_privatizationKernel(const float* A, const float* B, float* C, int N) {
    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    
    float results[REG_TILE_SIZE] = {0.0f};

    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        // Load data into shared memory
        if (row < N && t * TILE_SIZE + threadIdx.x < N) {
            As[threadIdx.y][threadIdx.x] = A[row * N + t * TILE_SIZE + threadIdx.x];
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0f;
        }

        for (int r = 0; r < REG_TILE_SIZE; ++r) {
            int b_row = t * TILE_SIZE + threadIdx.y;
            int b_col = col + r * TILE_SIZE;
            if (b_row < N && b_col < N) {
                Bs[threadIdx.y][threadIdx.x] = B[b_row * N + b_col];
            } else {
                Bs[threadIdx.y][threadIdx.x] = 0.0f;
            }

            __syncthreads();

            for (int k = 0; k < TILE_SIZE; ++k) {
                results[r] += As[threadIdx.y][k] * Bs[k][threadIdx.x];
            }

            __syncthreads();
        }
    }

    // Write results
    for (int r = 0; r < REG_TILE_SIZE; ++r) {
        int out_col = col + r * TILE_SIZE;
        if (row < N && out_col < N) {
            C[row * N + out_col] = results[r];
        }
    }
}

void runV5Privatization(const float* d_A, const float* d_B, float* d_C, int N, int blockSize) {
    // Always use TILE_SIZE for this kernel, ignore the blockSize parameter
    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 blocksPerGrid((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);
    V5_privatizationKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
}